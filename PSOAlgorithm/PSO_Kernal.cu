#include "hip/hip_runtime.h"
#include "Tools.cu"
#include "APoint.h"
#include "DevicePara.h"
#include "PSO.h"


extern "C" void InitPointDirectArray(int* pointDirectArray)
{
	InitPointDirectArray_Kernal<<<1,1>>>(pointDirectArray);
}
__global__ void InitPointDirectArray_Kernal(int* pointDirectArray)
{
	pointDirectArray[0] = -1;
	pointDirectArray[1] = -1;
	pointDirectArray[2] = -1;
	pointDirectArray[3] = -1;
	pointDirectArray[4] = -1;

	pointDirectArray[5] = -1;
	pointDirectArray[6] = 1;
	pointDirectArray[7] = 2;
	pointDirectArray[8] = 3;
	pointDirectArray[9] = 4;

	pointDirectArray[10] = -1;
	pointDirectArray[11] = 5;
	pointDirectArray[12] = 6;
	pointDirectArray[13] = 7;
	pointDirectArray[14] = 8;

	pointDirectArray[15] = -1;
	pointDirectArray[16] = 9;
	pointDirectArray[17] = 10;
	pointDirectArray[18] = 11;
	pointDirectArray[19] = 12;

	pointDirectArray[20] = -1;
	pointDirectArray[21] = 13;
	pointDirectArray[22] = 14;
	pointDirectArray[23] = 15;
	pointDirectArray[24] = 16;
}



//��ʼ��curBestPath_FitnessVal��ֵ 1
extern "C" void InitCurBestPathFit(double* curBestPath_FitnessVal)
{
	InitCurBestPathFit_Kernal<<<1,1>>>(curBestPath_FitnessVal);
}
__global__ void InitCurBestPathFit_Kernal(double* curBestPath_FitnessVal)
{
	curBestPath_FitnessVal[0] = 1000000;
}




//���µ�����ĵ�ǰ���Ž���Ϣ 1
extern "C" void UpdateCurBestPathInfo(int fitnessCount, double* fitness_GPU, int* bestParticleIndex,
	int tempStrConveyorList_PointSum, int tempCurveConveyorList_PointSum,
	/*���е���������Ϣ*/
	double* curBestFitnessVal, int inoutPSize, InoutPoint* inoutPoints, StraightConveyorInfo* strConveyorList,
	int* strConveyorListSum, Vector2Int* curveConveyorList, int* curveConveyorListSum,
	/*��ѵ���������Ϣ*/
	double* curBestPath_FitnessVal, int curBestPath_InoutPSize, InoutPoint* curBestPath_InoutPoints, StraightConveyorInfo* curBestPath_StrConveyorList,
	int* curBestPath_StrConveyorListSum, Vector2Int* curBestPath_CurveConveyorList, int* curBestPath_CurveConveyorListSum)
{
	UpdateCurBestPathInfo_Kernal <<<1, 1>>> (fitnessCount, fitness_GPU, bestParticleIndex,
		tempStrConveyorList_PointSum, tempCurveConveyorList_PointSum,
		/*���е���������Ϣ*/
		curBestFitnessVal, inoutPSize, inoutPoints, strConveyorList,
		strConveyorListSum, curveConveyorList, curveConveyorListSum,
		/*��ѵ���������Ϣ*/
		curBestPath_FitnessVal, curBestPath_InoutPSize, curBestPath_InoutPoints, curBestPath_StrConveyorList,
		curBestPath_StrConveyorListSum, curBestPath_CurveConveyorList, curBestPath_CurveConveyorListSum);
}
//���µ�ǰ���������·��Ϣ(��FitnessFunc�е���),ֻ�����һ��
__global__ void UpdateCurBestPathInfo_Kernal(int fitnessCount, double* fitness_GPU, int* bestParticleIndex,
	int tempStrConveyorList_PointSum, int tempCurveConveyorList_PointSum,
	/*���е���������Ϣ*/
	double* curBestFitnessVal, int inoutPSize, InoutPoint* inoutPoints, StraightConveyorInfo* strConveyorList,
	int* strConveyorListSum, Vector2Int* curveConveyorList, int* curveConveyorListSum,
	/*��ѵ���������Ϣ*/
	double* curBestPath_FitnessVal, int curBestPath_InoutPSize, InoutPoint* curBestPath_InoutPoints, StraightConveyorInfo* curBestPath_StrConveyorList,
	int* curBestPath_StrConveyorListSum, Vector2Int* curBestPath_CurveConveyorList, int* curBestPath_CurveConveyorListSum)
{
	//��curbest�����е���������ѱȽϣ�����curBest
	//���Ӧ��ֻҪ����һ��
	//ת������ֻ���ø��Ƶķ�����
	int bestIndex = bestParticleIndex[0];//������Ӷ�Ӧ���±�
	if (curBestPath_FitnessVal[0] > fitness_GPU[bestIndex * fitnessCount + 0]) {
		curBestPath_FitnessVal[0] = fitness_GPU[bestIndex * fitnessCount + 0];//ע��ƫ��ֵ
		for (int i = 0; i < curBestPath_InoutPSize; i++)
		{
			curBestPath_InoutPoints[i] = inoutPoints[bestIndex * inoutPSize + i];
		}
		for (int i = 0; i < strConveyorListSum[bestIndex]; i++) {
			curBestPath_StrConveyorList[i] = strConveyorList[bestIndex * tempStrConveyorList_PointSum + i];
		}
		curBestPath_StrConveyorListSum[0] = strConveyorListSum[bestIndex];
		for (int i = 0; i < curveConveyorListSum[bestIndex]; i++) {
			curBestPath_CurveConveyorList[i] = curveConveyorList[bestIndex * tempCurveConveyorList_PointSum + i];
		}
		curBestPath_CurveConveyorListSum[0] = curveConveyorListSum[bestIndex];
	}
}







//��������&������Ӧ��
extern "C" void UpdateParticle(int blockSum, int threadsPerBlock, int curIterNum, int maxIterNum, int dim, int fitnessCount, double w_, double C1_, double C2_, double dt_,
	/*�������Particle Particle* particles_,*/double* fitness_GPU, double* position_GPU, double* velocity_GPU, double* best_position_GPU, double* best_fitness_GPU,
	hiprandState* globalState, double* randomNumList, double* range_interval_, double* upper_bound_, double* lower_bound_, double* all_best_position_,
	/*ProblemParas problemParas*/Vector2* size, double* spaceLength, double workShopLength, double workShopWidth)
{
	UpdateParticle_Kernal << <blockSum, threadsPerBlock >> > (curIterNum, maxIterNum, dim, fitnessCount, w_, C1_, C2_, dt_,
		/*�������Particle Particle* particles_,*/fitness_GPU, position_GPU, velocity_GPU, best_position_GPU, best_fitness_GPU,
		globalState, randomNumList, range_interval_, upper_bound_, lower_bound_, all_best_position_,
		/*ProblemParas problemParas*/size, spaceLength, workShopLength, workShopWidth);
}
__global__ void UpdateParticle_Kernal(int curIterNum, int maxIterNum, int dim, int fitnessCount, double w_, double C1_, double C2_, double dt_,
	/*�������Particle Particle* particles_,*/double* fitness_GPU, double* position_GPU, double* velocity_GPU, double* best_position_GPU, double* best_fitness_GPU,
	hiprandState* globalState, double* randomNumList, double* range_interval_, double* upper_bound_, double* lower_bound_, double* all_best_position_,
	/*ProblemParas problemParas*/Vector2* size, double* spaceLength, double workShopLength, double workShopWidth)
{

	//���ӵ��±�i��Ҫ�Լ�����
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//�ȸ��³���Ȼ����ݳ���������ӵķ�Χ
	for (int j = 2; j < dim; j += 3)
	{
		double last_position = position_GPU[i * dim + j];

		velocity_GPU[i * dim + j] = w_ * velocity_GPU[i * dim + j] +
			C1_ * createARandomNum(globalState, i) * (best_position_GPU[i * dim + j] - position_GPU[i * dim + j]) +
			C2_ * createARandomNum(globalState, i) * (all_best_position_[i * dim + j] - position_GPU[i * dim + j]);
		position_GPU[i * dim + j] += dt_ * velocity_GPU[i * dim + j];

		// �����������������������
		if (upper_bound_ && lower_bound_)
		{
			if (position_GPU[i * dim + j] >= upper_bound_[j])//ע������豸����=Ҳ����
			{
				double thre = createARandomNum(globalState, i);//ֱ������һ�������
				if (last_position >= upper_bound_[j] - 1)//ע��upper_bound_[j]-1=3
				{
					position_GPU[i * dim + j] = createARandomNum(globalState, i) * range_interval_[j] + lower_bound_[j];
				}
				else if (thre < 0.5)
				{
					position_GPU[i * dim + j] = upper_bound_[j] - (upper_bound_[j] - last_position) * createARandomNum(globalState, i);
				}
				else
				{
					position_GPU[i * dim + j] = upper_bound_[j] - 0.5;
				}
			}
			if (position_GPU[i * dim + j] < lower_bound_[j])
			{
				double thre = createARandomNum(globalState, i);
				if (last_position == lower_bound_[j])
				{
					position_GPU[i * dim + j] = createARandomNum(globalState, i) * range_interval_[j] + lower_bound_[j];
				}
				else if (thre < 0.5)
				{
					position_GPU[i * dim + j] = lower_bound_[j] + (last_position - lower_bound_[j]) * createARandomNum(globalState, i);
				}
				else
				{
					position_GPU[i * dim + j] = lower_bound_[j];
				}
			}
		}
	}
	//���ݳ����޸��豸���½緶Χ
	for (int j = 2; j < dim; j += 3)
	{
		//doubleתint��ת��ΪDirection��Ȼ����ݳ������¼����豸�ߴ�ͳ����
		//Rotate90����Rotate270,�޸�������
		DeviceDirect curDirect = (DeviceDirect)(int)position_GPU[i * dim + j];
		if (curDirect == DeviceDirect::Rotate90 || curDirect == DeviceDirect::Rotate270)//��һ���ֿ���ҲҪ�ģ�enum��C++�﷨��
		{
			//x��y
			lower_bound_[j - 2] = 0 + size[j / 3].y * 0.5 + spaceLength[j / 3];
			lower_bound_[j - 1] = 0 + size[j / 3].x * 0.5 + spaceLength[j / 3];

			upper_bound_[j - 2] = workShopLength - size[j / 3].y * 0.5 - spaceLength[j / 3];
			upper_bound_[j - 1] = workShopWidth - size[j / 3].x * 0.5 - spaceLength[j / 3];

		}
		else
		{
			//x��y
			lower_bound_[j - 2] = 0 + size[j / 3].x * 0.5 + spaceLength[j / 3];
			lower_bound_[j - 1] = 0 + size[j / 3].y * 0.5 + spaceLength[j / 3];

			upper_bound_[j - 2] = workShopLength - size[j / 3].x * 0.5 - spaceLength[j / 3];
			upper_bound_[j - 1] = workShopWidth - size[j / 3].y * 0.5 - spaceLength[j / 3];

		}
		range_interval_[j - 2] = upper_bound_[j - 2] - lower_bound_[j - 2];
		range_interval_[j - 1] = upper_bound_[j - 1] - lower_bound_[j - 1];
	}
	//cout << endl;
	for (int j = 0; j < dim; j++)
	{
		if (j % 3 != 2)
		{
			//������һ�ε��������position��velocity
			double last_position = position_GPU[i * dim + j];

			velocity_GPU[i * dim + j] = w_ * velocity_GPU[i * dim + j] +
				C1_ * createARandomNum(globalState, i) * (best_position_GPU[i * dim + j] - position_GPU[i * dim + j]) +
				C2_ * createARandomNum(globalState, i) * (all_best_position_[i * dim + j] - position_GPU[i * dim + j]);
			position_GPU[i * dim + j] += dt_ * velocity_GPU[i * dim + j];

			// �����������������������
			if (upper_bound_ && lower_bound_)
			{
				if (position_GPU[i * dim + j] > upper_bound_[j])
				{
					double thre = createARandomNum(globalState, i);
					if (last_position >= upper_bound_[j])
					{
						position_GPU[i * dim + j] = createARandomNum(globalState, i) * range_interval_[j] + lower_bound_[j];
					}
					else if (thre < 0.5)
					{
						position_GPU[i * dim + j] = upper_bound_[j] - abs(upper_bound_[j] - last_position) * createARandomNum(globalState, i);
					}
					else
					{
						position_GPU[i * dim + j] = upper_bound_[j];
					}
				}
				if (position_GPU[i * dim + j] < lower_bound_[j])
				{
					double thre = createARandomNum(globalState, i);
					if (last_position <= lower_bound_[j])
					{
						position_GPU[i * dim + j] = createARandomNum(globalState, i) * range_interval_[j] + lower_bound_[j];
					}
					else if (thre < 0.5)
					{
						position_GPU[i * dim + j] = lower_bound_[j] + abs(last_position - lower_bound_[j]) * createARandomNum(globalState, i);
					}
					else
					{
						position_GPU[i * dim + j] = lower_bound_[j];
					}
				}
			}
		}

	}
}





//����Pbest��GPU����
extern "C" UpdatePbest_GPU(int blockSum, int threadsPerBlock, int dim_, int fitness_count, double* fitness_GPU, double* position_GPU,
	double* velocity_GPU, double* best_position_GPU, double* best_fitness_GPU, hiprandState* globalState)
{
	UpdatePbest_Kernal<<<blockSum>>>
}
__global__ void UpdatePbest_Kernal(int dim_, int fitness_count, double* fitness_GPU, double* position_GPU,
	double* velocity_GPU, double* best_position_GPU, double* best_fitness_GPU, hiprandState* globalState)
{
	//i��Ҫ����
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//�Ƚ���ʷpbest�͵�ǰ��Ӧ�ȣ������Ƿ�Ҫ����
	if (ComparePbest(i, fitness_count, fitness_GPU + i * fitness_count, best_fitness_GPU + i * fitness_count, globalState));
	{
		for (int j = 0; j < fitness_count; j++)
		{
			best_fitness_GPU[i * fitness_count + j] = fitness_GPU[i * fitness_count + j];
			//particles_[i].best_fitness_[j] = particles_[i].fitness_[j];
		}
		for (int j = 0; j < dim_; j++)
		{
			best_position_GPU[i * dim_ + j] = position_GPU[i * dim_ + j];
			//particles_[i].best_position_[j] = particles_[i].position_[j];
		}
	}
}





//����Gbest��GPU����
__global__ void UpdateGbest_Kernal(int fitness_count, int dim_, double* all_best_fitness_, double* all_best_position_, Particle* gbestList)
{
	//�±��Լ���
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//����
	for (int j = 0; j < fitness_count; j++)
	{
		all_best_fitness_[i * fitness_count + j] = gbestList[i].best_fitness_[j];
	}
	for (int k = 0; k < dim_; k++)
	{
		all_best_position_[i * dim_ + k] = gbestList[i].best_position_[k];
	}
}




// �Ƚ��������ӵ���Ӧ�ȣ��ж��Ƿ���ȫ֧�䣬�Ӷ������pbest
__device__ bool ComparePbest(int index, int fitness_count, double* fitness, double* pbestFitness, hiprandState* globalState)
{
	int numGreater = 0;
	int numLess = 0;
	for (int i = 0; i < fitness_count; i++)
	{
		if (fitness[i] < pbestFitness[i])
		{
			numGreater++;
		}
		if (fitness[i] > pbestFitness[i])
		{
			numLess++;
		}
	}
	//�����ǰ֧����ʷ������
	if (numGreater > 0 && numLess == 0)
	{
		return true;
	}
	//�����ʷ֧�䵱ǰ���ӣ�������
	else if (numGreater == 0 && numLess > 0)
	{
		return false;
	}
	//�������֧�䣬���ѡ����Ӧ��1�ĸ��ʸߵ㣩
	else
	{
		double randomProb = createARandomNum(globalState, index);//�������С��
		if (fitness[0] < pbestFitness[0])
		{
			if (randomProb < 0.75)
			{
				return true;
			}
			else
			{
				return false;
			}
		}
		else
		{
			if (randomProb > 0.5)
			{
				return true;
			}
			else
			{
				return false;
			}
		}
	}
}
